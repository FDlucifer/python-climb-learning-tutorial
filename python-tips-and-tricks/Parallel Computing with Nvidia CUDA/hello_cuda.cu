#include "hip/hip_runtime.h"
// nvcc hello_cuda.cu -o hello_cuda

#include <stdio.h>

__global__ void hello_cuda() {
    printf("hello cuda\n");
    printf("Block Index x: %d, block index y: %d, thread index x: %d, thread index y: %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y)
}

int main(int argc, char **argv) {
    hello_cuda<<<2,2>>>();
    hipDeviceSynchronize();

    return 0;
}
