#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void matrix_vector_product(float *A, float *v1, float *v2, int matrix_size) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (col == 0 && row < matrix_size) {
        float sum = 0.0f;
        for (int i = 0; i < matrix_size; i++) {
            sum += A[row * matrix_size + i] * v1[i];
        }
        v2[row] = sum;
    }
}

int main(int argc, char **argv) {
    float *A, *A_gpu;
    float *v1, *v1_gpu;
    float *v2, *v2_gpu;

    int matrix_size = 3;

    dim3 block_shape = dim3(32,32);

    dim3 grid_shape = dim3(max(1.0, ceil((float) matrix_size / (float) block_shape.x)),
                           max(1.0, ceil((float) matrix_size / (float) block_shape.y)));

    A = (float *) malloc(matrix_size * matrix_size * sizeof(float));
    v1 = (float *) malloc(matrix_size * sizeof(float));
    v2 = (float *) malloc(matrix_size * sizeof(float));

    for(int i = 0; i < matrix_size; i++) {
        for(int j = 0; j < matrix_size; j++) {
            A[i * matrix_size + j] = (float) i * matrix_size + j;
        }
    }

    for(int i = 0; i < matrix_size; i++) {
        v1[i] = (float) i;
    }

    hipMalloc((void **) &A_gpu, matrix_size * matrix_size * sizeof(float));
    hipMalloc((void **) &v1_gpu, matrix_size * sizeof(float));
    hipMalloc((void **) &v2_gpu, matrix_size * sizeof(float));

    hipMemcpy(A_gpu, A, matrix_size * matrix_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v1_gpu, v1, matrix_size * sizeof(float), hipMemcpyHostToDevice);

    matrix_vector_product<<<grid_shape, block_shape>>>(A_gpu, v1_gpu, v2_gpu, matrix_size);

    hipMemcpy(v2, v2_gpu, matrix_size * sizeof(float), hipMemcpyHostToDevice);

    for(int i = 0; i < matrix_size; i++) {
        printf("%.2f\n", v2[i]);
    }

    free(A);
    free(v1);
    free(v2);

    hipFree(A_gpu);
    hipFree(v1_gpu);
    hipFree(v2_gpu);

    return 0;
}
